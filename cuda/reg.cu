#include "hip/hip_runtime.h"
#include "TIPL/tipl.hpp"
#include "TIPL/cuda/mem.hpp"
#include "TIPL/cuda/basic_image.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void cuda_test(){
    ;
}

bool check_cuda(std::string& error_msg)
{
    int nDevices,Ver;
    if(hipGetDeviceCount(&nDevices) != hipSuccess ||
       hipDriverGetVersion(&Ver) != hipSuccess)
    {
        error_msg = "Cannot obtain GPU driver and device information. Please install a Nvidia driver";
        return false;
    }

    std::cout << "Device Count:" << nDevices << std::endl;
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) != hipSuccess)
        {
            error_msg = "Cannot obtain device information. Please update Nvidia driver";
            return false;
        }
        auto arch = prop.major*10+prop.minor;
        std::cout << "Device Number: " << i << std::endl;
        std::cout << "  Arch: " << arch << std::endl;
        std::cout << "  Device name: " << prop.name << std::endl;
        std::cout << "  Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
        std::cout << "  Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        std::cout << "  Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl;

        if(i == 0 && arch != CUDA_ARCH)
        {
            error_msg = "Incorrect CUDA Architecture. Please use DSI Studio (CUDA SM";
            error_msg += std::to_string(arch);
            error_msg += ").";
            return false;
        }
    }

    std::cout << "Driver Version: " << Ver << " DSI Studio CUDA Version: " << CUDART_VERSION << std::endl;
    cuda_test<<<1,1>>>();
    if(hipPeekAtLastError() != hipSuccess)
    {
        error_msg = "Failed to lauch cuda kernel:";
        error_msg += hipGetErrorName(hipGetLastError());
        error_msg += ". Please update Nvidia driver.";
        return false;
    }

    return true;
}

void cdm2_cuda(const tipl::image<3>& It,
               const tipl::image<3>& It2,
               const tipl::image<3>& Is,
               const tipl::image<3>& Is2,
               tipl::image<3,tipl::vector<3> >& d,
               tipl::image<3,tipl::vector<3> >& inv_d,
               bool& terminated,
               tipl::reg::cdm_param param)
{
    std::cout << "normalization using GPU" << std::endl;
    tipl::device_image<3> dIt(It),dIt2(It2),dIs(Is),dIs2(Is2);
    tipl::device_image<3,tipl::vector<3> > dd(It.shape()),inv_dd(It.shape());
    try{
        tipl::reg::cdm2_cuda(dIt,dIt2,dIs,dIs2,dd,inv_dd,terminated,param);
    }
    catch(std::runtime_error& er)
    {
        std::cout << "ERROR: " << er.what() << std::endl;
        std::cout << "switch to CPU" << std::endl;
        tipl::reg::cdm2(It,It2,Is,Is2,d,inv_d,terminated,param);
        return;
    }
    d.resize(It.shape());
    dd.vector().copy_to(d);
    inv_d.resize(It.shape());
    inv_dd.vector().copy_to(inv_d);

    hipDeviceSynchronize();

}

size_t linear_cuda(const tipl::image<3,float>& from,
                              tipl::vector<3> from_vs,
                              const tipl::image<3,float>& to,
                              tipl::vector<3> to_vs,
                              tipl::affine_transform<float>& arg,
                              tipl::reg::reg_type reg_type,
                              bool& terminated,
                              const float* bound)
{
    return tipl::reg::linear_two_way<tipl::reg::mutual_information_cuda>(from,from_vs,to,to_vs,arg,reg_type,[&](void){return terminated;},bound);
}



